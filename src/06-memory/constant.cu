
#include <hip/hip_runtime.h>
// test const memory via func parameter
__global__ void test_const_memory_via_func_param(float *out, const float *in)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    out[idx] = in[idx];
}

