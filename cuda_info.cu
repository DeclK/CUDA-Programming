
#include <hip/hip_runtime.h>
#include <iostream>

int main(){
  int dev = 0;
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, dev);
  std::cout << "GPU Device" << dev << ": " << devProp.name << std::endl;
  std::cout << "Num SM: " << devProp.multiProcessorCount << std::endl;
  std::cout << "Shared Memory Per Block: " << devProp.sharedMemPerBlock / 1024 << " KB" << std::endl;
  std::cout << "Max Threads Per Block: " << devProp.maxThreadsPerBlock << std::endl;
  std::cout << "Max Threads Per SM: " << devProp.maxThreadsPerMultiProcessor << std::endl;
  std::cout << "Max Warps Per SM: " << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
}